#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernelEfficientScanUpSweep(int array_length, int offset, int *data) {
          int index = (blockIdx.x * blockDim.x) + threadIdx.x;
          if (index >= array_length) {
            return;
          }

          int next_offset = offset * 2;

          // we only want to sum with threads that affect our next iteration
          if (index % next_offset == 0 && index + next_offset <= array_length) {
            data[index + next_offset - 1] += data[index + offset - 1];
          }
        }

        __global__ void kernelEfficientScanDownSweep(int array_length, int offset, int *data) {
          int index = (blockIdx.x * blockDim.x) + threadIdx.x;
          if (index >= array_length) {
            return;
          }

          int next_offset = 2 * offset;

          // we only want to work with threads that affect our next iteration
          if (index % next_offset == 0 && index + next_offset <= array_length) {
            int a_index = index + offset - 1;
            int b_index = index + next_offset - 1;

            int temp = data[a_index];
            data[a_index] = data[b_index];
            data[b_index] += temp;
          }
        }

        /**
        * Performs prefix-sum (aka scan) on idata, storing the result into odata.
        */

        // writing this here so that i can call it in compact and 
        // avoid the timer conflict issue
        void runScan(int n, int *odata, const int *idata) {
          int max_passes = ilog2ceil(n);
          int upper_bound = 1 << max_passes;

          dim3 blocksPerGrid((upper_bound + blockSize - 1) / blockSize);
          dim3 threadsPerBlock(blockSize);

          // array used for in-place threaded manipulations
          int *dev_temp;
          hipMalloc((void**)&dev_temp, sizeof(int) * upper_bound);
          // zero out array so also zeroing the unneeded elements past length n --> upper_bound
          hipMemset(dev_temp, 0, sizeof(int) * upper_bound);

          hipMemcpy(dev_temp, idata, sizeof(int) * n, hipMemcpyHostToDevice);

          // BEGIN: efficient scan upsweep
          for (int on_pass = 1; on_pass < upper_bound; on_pass *= 2) {
            kernelEfficientScanUpSweep << <blocksPerGrid, threadsPerBlock >> > (upper_bound, on_pass, dev_temp);
            checkCUDAError("kernelEfficientScanUpSweep failed!", __LINE__);
          }
          // END: efficient scan upsweep

          // BEGIN: efficient scan downsweep
          // set max element from upsweep iteration, dev_temp[n - 1], to 0
          hipMemset(dev_temp + upper_bound - 1, 0, sizeof(int));
          checkCUDAError("copying zero failed!", __LINE__);

          for (int pass_iteration = upper_bound / 2; pass_iteration > 0; pass_iteration /= 2) {
            kernelEfficientScanDownSweep << <blocksPerGrid, threadsPerBlock >> > (upper_bound, pass_iteration, dev_temp);
            checkCUDAError("kernelEfficientScanUpSweep failed!", __LINE__);
          }
          // END: efficient scan downsweep

          hipMemcpy(odata, dev_temp, sizeof(int)*n, hipMemcpyDeviceToHost);
          checkCUDAError("hipMemcpy dev_swapA and dev_swapB failed!", __LINE__);

          hipFree(dev_temp);
        }

        void scan(int n, int *odata, const int *idata) {
          //-----START
          timer().startGpuTimer();
          runScan(n, odata, idata);
          timer().endGpuTimer();
          //-----END
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            dim3 blocksPerGrid((n + blockSize - 1) / blockSize);
            dim3 threadsPerBlock(blockSize);

            int* data;
            int* final_data;
            int* bools;
            int* scan;
            hipMalloc((void**)&data, sizeof(int) * n);
            hipMalloc((void**)&final_data, sizeof(int) * n);
            hipMalloc((void**)&bools, sizeof(int) * n);
            hipMalloc((void**)&scan, sizeof(int) * n);
            checkCUDAError("mallocing failed!", __LINE__);

            hipMemcpy(data, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("copy to gpu data array failed!", __LINE__);

            //-----START
            timer().startGpuTimer();
            StreamCompaction::Common::kernMapToBoolean << <blocksPerGrid, threadsPerBlock>> > (n, bools, data);
            checkCUDAError("kernMapToBoolean failed!", __LINE__);

            StreamCompaction::Efficient::runScan(n, scan, bools);

            int last_value[1];
            hipMemcpy(&last_value, scan + n - 1, sizeof(int), hipMemcpyDeviceToHost);
            int count = (idata[n - 1] == 0) ? last_value[0] : last_value[0] + 1;

            StreamCompaction::Common::kernScatter << <blocksPerGrid, threadsPerBlock>> > (n, final_data, data, bools, scan);
            checkCUDAError("kernScatter failed!", __LINE__);
            timer().endGpuTimer();
            //-----END

            hipMemcpy(odata, final_data, sizeof(int) * n, hipMemcpyDeviceToHost);
            checkCUDAError("copy from gpu data array failed!", __LINE__);

            hipFree(data);
            hipFree(final_data);
            hipFree(bools);
            hipFree(scan);

            return count;
        }
    }

    namespace Radix {

      /*
      * Same idea as StreamCompaction::Radix except instead returns the opposite
      * boolean as the other function.
      */
      __global__ void kernMapToSpecifiedDigit(int n, const int binary_digit, int *opp_binary, const int *idata) {
        int index = (blockIdx.x * blockDim.x) + threadIdx.x;
        if (index >= n) {
          return;
        }

        // we want opp boolean of the binary masking's output
        opp_binary[index] = (idata[index] & binary_digit) ? 0 : 1;
      }

      /*
      * Implementing intermediary steps of gpu gems 3 section 39.3.3 for radix sort
      * using t and d values to create final index outputs to be used in scatter
      * to move all the elements to their final sorted ordering for the current digit
      * iteration
      */
      __global__ void kernResolveScatterInput(int n, int *scatter_input, const int num_false, const int* opp_binary, const int *scan_output) {
        int index = (blockIdx.x * blockDim.x) + threadIdx.x;
        if (index >= n) {
          return;
        }

        // t = i - f + num_false

        // b = !digits since i set digits to be the opposite of the normal binary rep for inputting to scan
        // d = b ? t : f
        //scatter_input[index] = index - scan_output[index] + num_false;

        scatter_input[index] = (!opp_binary[index]) ? index - scan_output[index] + num_false : scan_output[index];
      }

      /*
      * Scatter where all bools are true
      */
      __global__ void kernTrueScatter(int n, int *odata, const int *idata, const int *indices) {
        int index = (blockIdx.x * blockDim.x) + threadIdx.x;
        if (index >= n) {
          return;
        }

        odata[indices[index]] = idata[index];
      }

      /*
      * The CUDA implementation of radix sort on integer inputs
      */
      void radix(int n, int *odata, const int *idata, const int max_digit) {
        int upper_bound = 1 << max_digit;


        dim3 blocksPerGrid((upper_bound + blockSize - 1) / blockSize);
        dim3 threadsPerBlock(blockSize);

        int* dev_opposite_binary;
        int* dev_scan_output;
        int* dev_idata;
        int* dev_scatter_indices_input;
        int* dev_odata;
        hipMalloc((void**)&dev_opposite_binary, sizeof(int) * upper_bound);
        hipMalloc((void**)&dev_scan_output, sizeof(int) * upper_bound);
        hipMalloc((void**)&dev_idata, sizeof(int) * upper_bound);
        hipMalloc((void**)&dev_scatter_indices_input, sizeof(int) * upper_bound);
        hipMalloc((void**)&dev_odata, sizeof(int) * upper_bound);
        checkCUDAError("mallocing failed!", __LINE__);

        hipMemset(dev_idata, 0, sizeof(int) * upper_bound);
        hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);

        //int i = 1;
        for (int i = 1; i <= max_digit; i *= 2) {
          // map to opposite binary
          kernMapToSpecifiedDigit << <blocksPerGrid, threadsPerBlock >> > (n, i, dev_opposite_binary, dev_idata);

          // scan
          StreamCompaction::Efficient::runScan(n, dev_scan_output, dev_opposite_binary);

          // interpret scan input
          int last_value[1];
          hipMemcpy(&last_value, dev_scan_output + n - 1, sizeof(int), hipMemcpyDeviceToHost);
          last_value[0] += (int)(last_value[0] != 0); //(int)(n % 2 == 0);
          kernResolveScatterInput << <blocksPerGrid, threadsPerBlock >> > (n, dev_scatter_indices_input, last_value[0], dev_opposite_binary, dev_scan_output);

          // perform scatter
          kernTrueScatter << <blocksPerGrid, threadsPerBlock >> >(n, dev_odata, dev_idata, dev_scatter_indices_input);
          hipMemcpy(dev_idata, dev_odata, sizeof(int) * n, hipMemcpyDeviceToDevice);
        }

        hipMemcpy(odata, dev_idata, sizeof(int) * n, hipMemcpyDeviceToHost);

        hipFree(dev_opposite_binary);
        hipFree(dev_scan_output);
        hipFree(dev_idata);
        hipFree(dev_scatter_indices_input);
        hipFree(dev_odata);
      }
    }
}

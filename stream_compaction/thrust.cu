#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();

            int* dev_input;
            int* dev_output;
            hipMalloc((void**)&dev_input, sizeof(int) * n);
            hipMalloc((void**)&dev_output, sizeof(int) * n);

            hipMemcpy(dev_input, idata, sizeof(int) * n, hipMemcpyHostToDevice);

            thrust::device_ptr<int> input(dev_input);
            thrust::device_ptr<int> output(dev_output);
            
            thrust::exclusive_scan(input, input + n, output);

            hipMemcpy(odata, dev_output, sizeof(int) * n, hipMemcpyDeviceToHost);

            timer().endGpuTimer();
        }
    }
}